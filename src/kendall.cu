#include "cuseful.h"
#include "R.h"
#include "kendall.h"
#include "hip/hiprtc.h"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

#define NUMTHREADS 16

void masterKendall(const float * x,  size_t nx, 
  const float * y, size_t ny,
  size_t sampleSize, double * results,
  const char * kernel_src)
{
	size_t 
		outputLength = nx * ny, outputBytes = outputLength*sizeof(double),
		xBytes = nx*sampleSize*sizeof(float), 
		yBytes = ny*sampleSize*sizeof(float); 
	float
		* gpux, * gpuy; 
	double
		* gpuResults;
	dim3
		grid(nx, ny), block(NUMTHREADS, NUMTHREADS);

	hipMalloc((void **)&gpux, xBytes);
	hipMalloc((void **)&gpuy, yBytes);
	checkCudaError("input vector space allocation");

	hipMemcpy(gpux, x, xBytes, hipMemcpyHostToDevice);
	hipMemcpy(gpuy, y, yBytes, hipMemcpyHostToDevice);
	checkCudaError("copying input vectors to gpu");

	hipMalloc((void **)&gpuResults, outputBytes);
	checkCudaError("allocation of space for result matrix");

  void *args[] =
    { &gpux
    , &nx
    , &gpuy
    , &ny
    , &sampleSize
    , &gpuResults
    };
  cudaCompileLaunch(kernel_src, "gpuKendall", args,
      grid, block);

  hipFree(gpux);
  hipFree(gpuy);
  hipMemcpy(results, gpuResults, outputBytes, hipMemcpyDeviceToHost);
  hipFree(gpuResults);
  checkCudaError("copying results from gpu and cleaning up");
}
