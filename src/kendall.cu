#include "cuseful.h"
#include "R.h"
#include "kendall.h"
#include "hip/hiprtc.h"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

#define NUMTHREADS 16

void masterKendall(const float * x,  size_t nx, 
  const float * y, size_t ny,
  size_t sampleSize, double * results,
  const char * kernel_src)
{
	size_t 
		outputLength = nx * ny, outputBytes = outputLength*sizeof(double),
		xBytes = nx*sampleSize*sizeof(float), 
		yBytes = ny*sampleSize*sizeof(float); 
	float
		* gpux, * gpuy; 
	double
		* gpuResults;
	dim3
		initGrid(nx, ny), initBlock(NUMTHREADS, NUMTHREADS);

	hipMalloc((void **)&gpux, xBytes);
	hipMalloc((void **)&gpuy, yBytes);
	checkCudaError("input vector space allocation");

	hipMemcpy(gpux, x, xBytes, hipMemcpyHostToDevice);
	hipMemcpy(gpuy, y, yBytes, hipMemcpyHostToDevice);
	checkCudaError("copying input vectors to gpu");

	hipMalloc((void **)&gpuResults, outputBytes);
	checkCudaError("allocation of space for result matrix");

  void *args[] =
    { &gpux
    , &nx
    , &gpuy
    , &ny
    , &sampleSize
    , &gpuResults
    };
  int
    gridDim[3] = {nx, ny, 1},
    blockDim[3] = {NUMTHREADS, NUMTHREADS, 1};
  cudaCompileLaunch(kernel_src, "gpuKendall", args,
      gridDim, blockDim); 

  hipFree(gpux);
  hipFree(gpuy);
  hipMemcpy(results, gpuResults, outputBytes, hipMemcpyDeviceToHost);
  hipFree(gpuResults);
  checkCudaError("copying results from gpu and cleaning up");
}
