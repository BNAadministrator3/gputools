#include "hip/hip_runtime.h"
#include "R.h"
#include "hip/hiprtc.h"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

// Obtain compilation log from the program.
void printCompileLog(hiprtcProgram &prog) {
  size_t logSize;
  NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
  char * log = new char[logSize];
  NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, log));
  warning(log);
  delete[] log;
}

void cudaCompileLaunch(const char * kernelSrc,
                       const char * kernelName,
                       void * args[],
                       const dim3 &gridDim, const dim3 &blockDim,
                       hipStream_t stream)
{
  hiprtcProgram prog;
  NVRTC_SAFE_CALL(
      hiprtcCreateProgram(&prog,  // prog
        kernelSrc,               // buffer
        kernelName,              // name
        0,                       // numHeaders
        NULL,                    // headers
        NULL));                  // includeNames

  hiprtcResult compileResult = hiprtcCompileProgram(prog, 0, NULL);
  if (compileResult != HIPRTC_SUCCESS) {
    printCompileLog(prog);
    error("\ncuda kernel compile failed");
  }

  //  Obtain PTX from the program.
  size_t ptxSize;
  NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));

  char * ptx = Calloc(ptxSize, char);
  NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx));

  //  Destroy the program.
  NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));

  //  Load the generated PTX and get a handle to the SAXPY kernel.
  CUDA_SAFE_CALL(hipInit(0));

  hipModule_t module;
  CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx, 0, 0, 0));

  hipFunction_t kernel;
  CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, kernelName));

  CUDA_SAFE_CALL(
    hipModuleLaunchKernel(kernel,
      gridDim.x, gridDim.y, gridDim.z,    // grid dim
      blockDim.x, blockDim.y, blockDim.z, // block dim
      0, stream,                    // shared mem and stream
      args, 0));                  // arguments
  CUDA_SAFE_CALL(hipCtxSynchronize());

  CUDA_SAFE_CALL(hipModuleUnload(module));
  Free(ptx);
}
