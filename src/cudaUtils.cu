#include "R.h"
#include "hip/hiprtc.h"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

void cudaCompileLaunch(const char * kernelSrc, const char * kernelName,
    void * args[], int gridDim[3], int blockDim[3])
{
  hiprtcProgram prog;
  NVRTC_SAFE_CALL(
      hiprtcCreateProgram(&prog,  // prog
        kernelSrc,               // buffer
        kernelName,              // name
        0,                       // numHeaders
        NULL,                    // headers
        NULL));                  // includeNames

  hiprtcResult compileResult = hiprtcCompileProgram(prog, 0, NULL);
  if (compileResult != HIPRTC_SUCCESS) error("cuda kernel compile failed");

  //  Obtain PTX from the program.
  size_t ptxSize;
  NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));

  char * ptx = Calloc(ptxSize, char);
  NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx));

  //  Destroy the program.
  NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));

  //  Load the generated PTX and get a handle to the SAXPY kernel.
  CUDA_SAFE_CALL(hipInit(0));

  hipModule_t module;
  CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx, 0, 0, 0));

  hipFunction_t kernel;
  CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, kernelName));

  CUDA_SAFE_CALL(
    hipModuleLaunchKernel(kernel,
      gridDim[0], gridDim[1], gridDim[2],    // grid dim
      blockDim[0], blockDim[1], blockDim[2], // block dim
      0, NULL,                    // shared mem and stream
      args, 0));                  // arguments
  CUDA_SAFE_CALL(hipCtxSynchronize());

  CUDA_SAFE_CALL(hipModuleUnload(module));
  Free(ptx);
}
