#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#include<hip/hip_runtime.h>
#include<hipblas.h>

#include<cuseful.h>

#include<R.h>
#include<Rinternals.h>
#include<matmult.h>

SEXP gpuMatMult(SEXP a, SEXP b) {
	double
        * xa = REAL(a), * xb = REAL(b),
		* gpua, * gpub, * gpuc;

    SEXP
        dima = getAttrib(a, R_DimSymbol),
        dimb = getAttrib(b, R_DimSymbol);

	int
		rowsa = INTEGER(dima)[0], colsa = INTEGER(dima)[1],
		rowsb = INTEGER(dimb)[0], colsb = INTEGER(dimb)[1];

	hipblasStatus_t stat;
	hipblasHandle_t handle;

	hipError_t cudaStat;

	cudaStat = hipMalloc((void**) &gpua, rowsa * colsa * sizeof(double));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return NULL;
	}  

	cudaStat = hipMalloc((void**) &gpub, rowsb * colsb * sizeof(double));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return NULL;
	}  

//	hipblasOperation_t opA = tpA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
//	hipblasOperation_t opB = tpB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
//
//	int rowsOpA = tpA ? colsa : rowsa;
//	int colsOpA = tpA ? rowsa : colsa;
//	int colsOpB = tpB ? rowsb : colsb;

	int
        rowsOpA = rowsa, colsOpA = colsa, colsOpB = colsb;

	cudaStat = hipMalloc((void**) &gpuc, rowsOpA * colsOpB * sizeof(double));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return NULL;
	}  

	stat = hipblasCreate(&handle);
	if(stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS initialization failed\n");
		return NULL;
	}

	stat = hipblasSetMatrix(rowsa, colsa, sizeof(double), xa, rowsa,
		gpua, rowsa);
	if(stat != HIPBLAS_STATUS_SUCCESS) {
		printf("data download failed\n");
		hipFree(gpuc);
		hipFree(gpub);
		hipFree(gpua);
		hipblasDestroy(handle);
		return NULL;
	}

	stat = hipblasSetMatrix(rowsb, colsb, sizeof(double), xb, rowsb,
		gpub, rowsb);
	if(stat != HIPBLAS_STATUS_SUCCESS) {
		printf("data download failed\n");
		hipFree(gpuc);
		hipFree(gpub);
		hipFree(gpua);
		hipblasDestroy(handle);
		return NULL;
	}

	const double alpha = 1.0, beta = 0.0;
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowsOpA, colsOpB, colsOpA, &alpha,
		(const double *) gpua, rowsa, (const double *) gpub, rowsb,
		&beta, gpuc, rowsOpA);

    SEXP ab, dimab;
    PROTECT(ab = allocVector(REALSXP, rowsOpA * colsOpB));
    PROTECT(dimab = allocVector(INTSXP, 2));
    INTEGER(dimab)[0] = rowsOpA; INTEGER(dimab)[1] = colsOpB;
    setAttrib(ab, R_DimSymbol, dimab);

    double * xab = REAL(ab);
	stat = hipblasGetMatrix(rowsOpA, colsOpB, sizeof(double), gpuc, rowsOpA,
		xab, rowsOpA);
	if(stat != HIPBLAS_STATUS_SUCCESS) {
		printf("data upload failed\n");
		hipFree(gpuc);
		hipFree(gpub);
		hipFree(gpua);
		hipblasDestroy(handle);
		return NULL;
	}

	hipFree(gpua);
	hipFree(gpub);
	hipFree(gpuc);

	hipblasDestroy(handle);
    UNPROTECT(2);
	return ab;
}
