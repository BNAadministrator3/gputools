
#include <hip/hip_runtime.h>
#define NUMTHREADS 16
#define THREADWORK 32

template<typename T>
__global__ void gpuKendall(const T * a, size_t na,
                           const T * b, size_t nb,
                           size_t sampleSize,
                           double * results) 
{
	size_t 
		i, j, tests, 
		tx = threadIdx.x, ty = threadIdx.y, 
		bx = blockIdx.x, by = blockIdx.y,
		rowa = bx * sampleSize, rowb = by * sampleSize;
	T 
		discordant, concordant = 0.0,
		numer, denom;

	__shared__ T threadSums[NUMTHREADS*NUMTHREADS];

	for(i = tx; i < sampleSize; i += NUMTHREADS) {
		for(j = i+1+ty; j < sampleSize; j += NUMTHREADS) {
			tests = ((a[rowa+j] >  a[rowa+i]) && (b[rowb+j] >  b[rowb+i]))
				+ ((a[rowa+j] <  a[rowa+i]) && (b[rowb+j] <  b[rowb+i])) 
				+ ((a[rowa+j] == a[rowa+i]) && (b[rowb+j] == b[rowb+i])); 
			concordant = concordant + (double) tests;
		}
	}
	threadSums[tx*NUMTHREADS+ty] = concordant;

	__syncthreads();
	for(i = NUMTHREADS >> 1; i > 0; i >>= 1) {
		if(ty < i)
			threadSums[tx*NUMTHREADS+ty] += threadSums[tx*NUMTHREADS+ty+i];
		__syncthreads();
	}
  for(i = NUMTHREADS >> 1; i > 0; i >>= 1) {
    if((tx < i) && (ty == 0))
      threadSums[tx*NUMTHREADS] += threadSums[(tx+i)*NUMTHREADS];
    __syncthreads();
  }

	if((tx == 0) && (ty == 0)) {
		concordant = threadSums[0];
		denom = (double) sampleSize;
		denom = (denom * (denom - 1.f)) / 2.f; discordant = denom - concordant;
		numer = concordant - discordant;
		results[by*na+bx] = ((double) numer) / ((double) denom);
	}
}
